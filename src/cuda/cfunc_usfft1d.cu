#include "hip/hip_runtime.h"
#include "kernels_usfft1d.cu"
#include "cfunc_usfft1d.cuh"
#define EPS 1e-3

cfunc_usfft1d::cfunc_usfft1d(size_t n0_, size_t n1_, size_t n2_, size_t deth_) {

  n0 = n2_; // reorder from python
  n1 = n1_;
  n2 = n0_;
  deth = deth_;

  mu2 = -log(EPS) / (2 * n2 * n2);
  m2 = ceil(2 * n2 * 1 / PI * sqrt(-mu2 * log(EPS) + (mu2 * n2) * (mu2 * n2) / 4));

  int ffts[1];
  int idist;
  int istride;
  int inembed[1];

  // cfunc_usfft1d 1d
  ffts[0] = 2 * n2;
  idist = 1; //(2*n0 + 2*m) * n2 * n1;
  istride = n1 * n0;
  inembed[0] = n1 * n0;
  
  hipMalloc((void **)&fdee1d, n1 * n0 * (2 * n2 + 2 * m2) * sizeof(float2));
  hipMalloc((void **)&x, deth * sizeof(float));
  hipfftPlanMany(&plan1dchunk, 1, ffts, inembed, istride, idist, inembed, istride, idist, HIPFFT_C2C, n0*n1);
  
  BS1d = dim3(16, 8, 8);
  GS1d0 = dim3(ceil(n0 / (float)BS1d.x), ceil(n1 / (float)BS1d.y), ceil(n2 / (float)BS1d.z));
  GS1d1 = dim3(ceil(n0 / (float)BS1d.x), ceil(n1 / (float)BS1d.y), ceil((2 * n2 + 2 * m2) / (float)BS1d.z));
  GS1d2 = dim3(ceil(n0 / (float)BS1d.x), ceil(n1 / (float)BS1d.y), ceil(deth / (float)BS1d.z));
  
  BS1dx = dim3(1024);
  GS1dx = dim3(ceil(deth/(float)BS1dx.x));
}

// destructor, memory deallocation
cfunc_usfft1d::~cfunc_usfft1d() { free(); }

void cfunc_usfft1d::free() {
  if (!is_free) {
    hipFree(fdee1d);
    hipfftDestroy(plan1dchunk);
    is_free = true;
  }
}

void cfunc_usfft1d::fwd(size_t g_, size_t f_, float phi, size_t stream_) {

  f = (float *)f_;
  g = (float2 *)g_;
  stream = (hipStream_t)stream_;    
  
  hipfftSetStream(plan1dchunk, stream);
  hipMemsetAsync(fdee1d, 0, n0 * n1 * (2 * n2 + 2 * m2) * sizeof(float2), stream);
  take_x<<<GS1dx, BS1dx, 0, stream>>>(x, phi, deth);
  divker1d<<<GS1d0, BS1d, 0, stream>>>(fdee1d, f, n0, n1, n2, m2, mu2, 0);
  fftshiftc1d<<<GS1d1, BS1d, 0, stream>>>(fdee1d, n0, n1, 2 * n2 + 2 * m2);
  hipfftExecC2C(plan1dchunk, (hipfftComplex *)&fdee1d[m2 * n0 * n1].x,
               (hipfftComplex *)&fdee1d[m2 * n0 * n1].x, HIPFFT_FORWARD);
  fftshiftc1d<<<GS1d1, BS1d, 0, stream>>>(fdee1d, n0, n1, 2 * n2 + 2 * m2);
  wrap1d<<<GS1d1, BS1d, 0, stream>>>(fdee1d, n0, n1, n2, m2, 0);  
  gather1d<<<GS1d2, BS1d, 0, stream>>>(g, fdee1d, x, m2, mu2, n0, n1, n2, deth, 0);
}

void cfunc_usfft1d::adj(size_t f_, size_t g_, float phi, size_t stream_) {

  f = (float *)f_;
  g = (float2 *)g_;
  stream = (hipStream_t)stream_;    
  
  hipfftSetStream(plan1dchunk, stream);
  hipMemsetAsync(fdee1d, 0, n0 * n1 * (2 * n2 + 2 * m2) * sizeof(float2), stream);
  take_x<<<GS1dx, BS1dx, 0, stream>>>(x, phi, deth);

  //could be done with r2x if needed
  gather1d<<<GS1d2, BS1d, 0, stream>>>(g, fdee1d, x, m2, mu2, n0, n1, n2, deth, 1);
  wrap1d<<<GS1d1, BS1d, 0, stream>>>(fdee1d, n0, n1, n2, m2, 1);   
  fftshiftc1d<<<GS1d1, BS1d, 0, stream>>>(fdee1d, n0, n1, 2 * n2 + 2 * m2);
  hipfftExecC2C(plan1dchunk, (hipfftComplex *)&fdee1d[m2 * n0 * n1].x,
               (hipfftComplex *)&fdee1d[m2 * n0 * n1].x, HIPFFT_BACKWARD);
  fftshiftc1d<<<GS1d1, BS1d, 0, stream>>>(fdee1d, n0, n1, 2 * n2 + 2 * m2);
  divker1d<<<GS1d0, BS1d, 0, stream>>>(fdee1d, f, n0, n1, n2, m2, mu2, 1);  
}
